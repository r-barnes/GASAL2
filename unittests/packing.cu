#include "doctest.h"
#include <kernels/pack_rc_seqs.cuh>

#include <string>
#include <unordered_map>
#include <vector>

/*
a=['A','G','C','T']
import random
''.join(random.choices(a,k=8))
*/

#define CHECKCUDAERROR(error) \
    {\
      const auto err=(error); \
      if (err!=hipSuccess) { \
        fprintf(stderr, "[GASAL CUDA ERROR:] %s(CUDA error no.=%d). Line no. %d in file %s\n", hipGetErrorString(err), err,  __LINE__, __FILE__); \
      }\
      REQUIRE(err==hipSuccess); \
    }

TEST_CASE("Packing"){
  const std::string unpacked_query_seed = "GAACTGCCGAGAAGTCACAGAAGGGACTGTGG";
  std::string unpacked_query;
  for(int i=0;i<100;i++)
    unpacked_query += unpacked_query_seed;

  CHECK(unpacked_query.size()%8==0);

  const auto unpacked_size = unpacked_query.size()/4;
  const auto packed_size   = unpacked_size/2;

  uint32_t *unpacked_query_dev;
  CHECKCUDAERROR(hipMalloc(&unpacked_query_dev, unpacked_size*sizeof(uint32_t)));
  CHECKCUDAERROR(hipMemcpy(unpacked_query_dev, (uint32_t*)unpacked_query.data(), unpacked_size*sizeof(uint32_t), hipMemcpyHostToDevice));

  uint32_t *packed_query_dev;
  CHECKCUDAERROR(hipMalloc(&packed_query_dev, packed_size*sizeof(uint32_t)));

  const uint32_t BLOCKDIM = 128;
  const uint32_t N_BLOCKS = 30;

  pack_data<<<N_BLOCKS, BLOCKDIM>>>(
    unpacked_query_dev,
    packed_query_dev,
    unpacked_size
  );

  const auto err = hipGetLastError();
  CHECK(err==hipSuccess);

  CHECKCUDAERROR(hipDeviceSynchronize());

  std::vector<uint32_t> packed_query (packed_size);

  CHECKCUDAERROR(hipMemcpy(packed_query.data(),  packed_query_dev,  packed_size*sizeof(uint32_t), hipMemcpyDeviceToHost));

  const std::unordered_map<int,char> trans{{1,'A'}, {3, 'C'}, {7, 'G'}, {4, 'T'}, {0, '-'}};

  std::string packed_result;
  for(const auto &x: packed_query){
    packed_result.push_back(trans.at((x>>28)&0xF));
    packed_result.push_back(trans.at((x>>24)&0xF));
    packed_result.push_back(trans.at((x>>20)&0xF));
    packed_result.push_back(trans.at((x>>16)&0xF));
    packed_result.push_back(trans.at((x>>12)&0xF));
    packed_result.push_back(trans.at((x>> 8)&0xF));
    packed_result.push_back(trans.at((x>> 4)&0xF));
    packed_result.push_back(trans.at((x>> 0)&0xF));
  }

  CHECK(unpacked_query==packed_result);
}



TEST_CASE("Simple complement"){
  uint32_t packed = 0;
  packed |= (('G'&0xF)<<28);
  packed |= (('C'&0xF)<<24);
  packed |= (('T'&0xF)<<20);
  packed |= (('T'&0xF)<<16);
  packed |= (('G'&0xF)<<12);
  packed |= (('T'&0xF)<< 8);
  packed |= (('A'&0xF)<< 4);
  packed |= (('A'&0xF)<< 0);

  const auto complement = packed_complement1(packed);

  CHECK( ((complement>>28)&0xF) == ('C' & 0xF));
  CHECK( ((complement>>24)&0xF) == ('G' & 0xF));
  CHECK( ((complement>>20)&0xF) == ('A' & 0xF));
  CHECK( ((complement>>16)&0xF) == ('A' & 0xF));
  CHECK( ((complement>>12)&0xF) == ('C' & 0xF));
  CHECK( ((complement>> 8)&0xF) == ('A' & 0xF));
  CHECK( ((complement>> 4)&0xF) == ('T' & 0xF));
  CHECK( ((complement>> 0)&0xF) == ('T' & 0xF));
}